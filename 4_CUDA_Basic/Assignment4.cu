#define BLOCK_DIM	4
#define TILE_DIM	BLOCK_DIM



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include<time.h>


void PrintMatrixToText(int* mtxArray, int height, int width, const char* fileName);
// Matrix Mult Kernel
__global__ void matrixMult( int* A, int* B, int* C, const int n, const int m, const int k, const int numPhase)
{
	__shared__ int Ads[TILE_DIM][TILE_DIM];
	__shared__ int Bds[TILE_DIM][TILE_DIM];
	/*Todo: Write C = A*B */
	int bx = blockIdx.x;  int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;

	int Row = by * blockDim.y + ty;
	int Col = bx * blockDim.x + tx;

	int Pvalue = 0;

	// Loop over the A and B tiles required to compute the C element
	for ( int ph=0 ; ph<numPhase ; ++ph )
	{
		// Collaborative loading of A and B tiles into shared memory
		if(  Row < n && ph*TILE_DIM + tx < m )	Ads[ty][tx] = A[ Row*m + ph*TILE_DIM + tx  ];
		else	Ads[ty][tx] = 0;
		
		if(  ph*TILE_DIM + ty < m && Col < k )	Bds[ty][tx] = B[ (ph*TILE_DIM + ty)*k + Col ];
		else	Bds[ty][tx] = 0;
		
		__syncthreads();

		if( Col < k && Row < n )
		{
			for ( int i=0 ; i<TILE_DIM ; ++i )
			{
				Pvalue += (Ads[ ty ][ i ] * Bds[ i ][ tx ]);//인덱스 입력
			}
			__syncthreads();
		}
	}

	if( Col < k && Row < n )	C[Row*k + Col] = Pvalue;
	
}

int main(){
	int *h_a, *h_b, *h_c;	// Host Variables
	int *d_a, *d_b, *d_c;	// Device Variables
	int n,m,k;
	
	
	// Receive Parameters
	/*Todo:Write Receive Parameters through console commands*/
	printf("Input the Value of n : ");
	scanf("%d",&n);
	printf("Input the Value of m : ");
	scanf("%d",&m);
	printf("Input the Value of k : ");
	scanf("%d",&k);	
	
	
	// Alloc & Initialize Host Input Matrices	
	h_a = (int*)malloc(n*m*sizeof(int));
	h_b = (int*)malloc(m*k*sizeof(int));
	h_c = (int*)malloc(n*k*sizeof(int));
	
	srand(time(NULL));
	
	/*Todo: Initialize A and B with Random Varialbles Ranged (-10, 10)*/
	for(int i = 0 ; i < n*m ; i++)
	{
		h_a[i] = rand() % 21 - 10;
	}
	for(int i = 0 ; i < m*k ; i++)
	{
		h_b[i] = rand() % 21 - 10;
	}
	
	// Allocate Device Memory
	hipMalloc((void **) &d_a, n*m*sizeof(int));
	hipMalloc((void **) &d_b, m*k*sizeof(int));
	hipMalloc((void **) &d_c, n*k*sizeof(int));

	// Copy Input Matrices to Device Memory
	hipMemcpy(d_a, h_a, n*m*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, m*k*sizeof(int), hipMemcpyHostToDevice);

	
	// Set Grid/Block Dimensions
	
	int dimxG = (int)ceil((float)k/(float)BLOCK_DIM);
	int dimyG = (int)ceil((float)n/(float)BLOCK_DIM);
	dim3 dimGrid( dimxG , dimyG);
	dim3 dimBlock(BLOCK_DIM,BLOCK_DIM);

	//페이징
	int numPhase = (int)ceil((float)m/(float)BLOCK_DIM);
		
	// Launch Kernel
	matrixMult<<<dimGrid ,dimBlock >>>(d_a, d_b, d_c, n, m, k, numPhase);

	// Copy Result to Host
	hipMemcpy( h_c, d_c, n*k*sizeof(int) , hipMemcpyDeviceToHost );
	
	// Free Device Memory
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	
	
	// Print Results
	PrintMatrixToText(h_a,n,m,"matrixA.txt");
	PrintMatrixToText(h_b,m,k,"matrixB.txt");
	PrintMatrixToText(h_c,n,k,"matrixC.txt");
	
	// Free Host Memory
	free(h_a); free(h_b); free(h_c);
	
	return 0;
}

// Utilities
void PrintMatrixToText(int* mtxArray, int height, int width, const char* fileName){
	FILE *f = fopen(fileName, "w");
	for ( int i=0 ; i<height ; i++ )
	{
		for ( int j=0 ; j<width ; j++ )
		{
			fprintf(f, "%d\t" , mtxArray[i*width+j] );
		}
		fprintf(f,"\n" );
	}	
	
	fclose(f);
}

