#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
using namespace std;

struct vec3
{
	float x,y,z;	
};

__global__ void vectorAdd(vec3 *c, vec3 *a, vec3 *b)
{
	int tid = threadIdx.x;
	/*	  1-1. write vector addition function						*/
	c->x = a->x + b->x;
	c->y = a->y + b->y;
	c->z = a->z + b->z;
	
}


int main( void )
{
	/*	  2-1. Check whether a proper device is mounted 			*/
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		cout << "hipSetDevice failed! Do you have a CUDA-capable GPU installed?" << endl;
	}
	
	/*	  2-2. Declare Host and Device pointer variables			*/
	/*    2-3. Allocate Host memory									*/
	struct vec3 *a= (vec3*)malloc(sizeof(vec3));
	struct vec3 *b= (vec3*)malloc(sizeof(vec3));
	struct vec3 *c= (vec3*)malloc(sizeof(vec3));
	struct vec3* dev_a;
	struct vec3* dev_b;
	struct vec3* dev_c;
	
	/*    2-4. Allocate Device memory								*/
	/*    2-5. Check that memory is allocated well on Device		*/
	cudaStatus = hipMalloc((void**)&dev_a, sizeof(vec3));
	if(cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&dev_b, sizeof(vec3));
	if(cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&dev_c, sizeof(vec3));
	if(cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}
	
	/*    2-6. Setup Input values to host array						*/
	a->x = 1;
	a->y = 2;
	a->z = 3;
	b->x = 10;
	b->y = 20;
	b->z = 30;
	
	/*    2-7. Copy memory for Input array from Host to Device		*/
	cudaStatus = hipMemcpy(dev_a, a, sizeof(vec3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}
	cudaStatus = hipMemcpy(dev_b, b, sizeof(vec3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}
	
	/*	  2-8. Call Kernel Function with <<<1, 1>>>					*/
	vectorAdd<<<1,1>>>(dev_c, dev_a, dev_b);

	/*    2-9. Copy memory for Result from Device to Host			*/
	cudaStatus = hipMemcpy(c, dev_c, sizeof(vec3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}
	
	/*    2-10. Print Results	*/
	printf("(x:%0.f, y:%0.f, z:%0.f) + ", a->x, b->y, b->z);
	printf("(x:%0.f, y:%0.f, z:%0.f)\n", b->x, b->y, b->z);
	printf("= (x:%0.f, y:%0.f, z:%0.f)\n", c->x, c->y, c->z);
	/*    2-11. Release Host and Device memory						*/
	hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
	
	return 0;

}

// nvcc [fileName] -o [outName]